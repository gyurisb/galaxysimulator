#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math.h>

#include "GalaxySimulator.h"

const int ThreadCount = 500;

__global__ void move(Body* space, Body* nextSpace, const int BodyCount)
{
	int i = blockIdx.x*ThreadCount + threadIdx.x;
	if (i >= BodyCount)
		return;

	Body* b2 = space + i;
	Body* b = nextSpace + i;
	double b2x = b2->x;
	double b2y = b2->y;
	double ax_sum = 0.0;
	double ay_sum = 0.0;
	for (int k = 0; k < BodyCount; k++)
	{
		Body* b1 = space + k;
		if (i != k)
		{
			double dx = b2x - b1->x;
			double dy = b2y - b1->y;
			double dist2 = dx*dx + dy*dy;
			double dist = sqrt(dist2);
			double a = (double)b1->mass / dist2;
			double ax = a * (dx / dist);
			double ay = a * (dy / dist);

			ax_sum += ax;
			ay_sum += ay;
		}
	}

	double ax = -GravitationalConstant * ax_sum;
	double ay = -GravitationalConstant * ay_sum;

	b->x = b2->x + b2->vx;
	b->y = b2->y + b2->vy;
	b->vx = b2->vx + ax;
	b->vy = b2->vy + ay;
	if (b->x <= -SpaceBorder || b->x >= SpaceBorder || b->y <= -SpaceBorder || b->y >= SpaceBorder)
		b->mass = -1;
	else
		b->mass = b2->mass;
}

Body *devInput = 0;
Body *devOutput = 0;

// Helper function for using CUDA to add vectors in parallel.
hipError_t moveWithCuda(Body* space, Body* nextSpace)
{
	hipError_t cudaStatus;
	
	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(devInput, space, BodyCount * sizeof(Body), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	const int BlockCount = ceil(BodyCount / (double)ThreadCount);
	// Launch a kernel on the GPU with one thread for each element.
	move<<<BlockCount, ThreadCount>>>(devInput, devOutput, BodyCount);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(nextSpace, devOutput, BodyCount * sizeof(Body), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	return cudaStatus;
}

class SimulateWithCudaStrategy
{
public:

	static void initialize(Body* space) {
	}

	static void simulateDay(Body* space, Body* nextSpace) {
		hipError_t cudaStatus = moveWithCuda(space, nextSpace);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "moveWithCuda failed!");
			throw "moveWithCuda failed!";
		}
	}
};

int main()
{
	// Choose which GPU to run on, change this on a multi-GPU system.
	hipError_t cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&devInput, BodyCount * sizeof(Body));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&devOutput, BodyCount * sizeof(Body));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	simulateGalaxy<SimulateWithCudaStrategy>();

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
		goto Error;
	}

Error:
	hipFree(devInput);
	hipFree(devOutput);
	return 0;
}
